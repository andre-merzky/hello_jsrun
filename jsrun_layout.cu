#include <stdio.h>
#include <mpi.h>
#include <sched.h>
#include <nvml.h>

int main(int argc, char *argv[]){

	MPI_Init(&argc, &argv);

	int size;
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	char name[MPI_MAX_PROCESSOR_NAME];
	int resultlength;
	MPI_Get_processor_name(name, &resultlength);

	// Find out which HWThread is being used
	int hwthread = sched_getcpu();

	// Find how many GPUs CUDA runtime says are available
	int num_devices = 0;
	hipGetDeviceCount(&num_devices);

	// Set output based on command line argument
	// => verbose shows BusID and UUID for GPUs
	char output_flag[64];
	strcpy(output_flag, "not_verbose");
	if(argc > 1){ 
		if(strlen(argv[1]) >= sizeof(output_flag)){
			printf("Argument too long: %s\n", argv[1]);
			exit(1);
		}
		else{
			strcpy(output_flag, argv[1]);
		}
	}

	if(num_devices == 0){
		printf("MPI Rank %03d of %03d on HWThread %03d of Node %s\n", rank, size, hwthread, name);
	}
	else{

		// NVML is needed to query the UUID of GPUs, which
		// allows us to check which GPU is actually being used
		// by each MPI rank
		nvmlInit();

		char uuid[NVML_DEVICE_UUID_BUFFER_SIZE];
		char busid[64];

		int gpu_id;

		// Loop over the GPUs available to each MPI rank
		for(int i=0; i<num_devices; i++){

			hipSetDevice(i);

			// Get the PCIBusId for each GPU and use it to query for UUID
			hipDeviceGetPCIBusId(busid, 64, i);

			// Get UUID for the device based on busid
			nvmlDevice_t device;
			nvmlDeviceGetHandleByPciBusId(busid, &device);
			nvmlDeviceGetUUID(device, uuid, NVML_DEVICE_UUID_BUFFER_SIZE);

			// Map DomainID and BusID to node-local GPU ID
			if(strcmp(busid, "0004:04:00.0") == 0){
				gpu_id = 0;
			}else if(strcmp(busid, "0004:05:00.0") == 0){
				gpu_id = 1;
			}else if(strcmp(busid, "0004:06:00.0") == 0){
				gpu_id = 2;
			}else if(strcmp(busid, "0035:03:00.0") == 0){
				gpu_id = 3;
			}else if(strcmp(busid, "0035:04:00.0") == 0){
				gpu_id = 4;
			}else if(strcmp(busid, "0035:05:00.0") == 0){
				gpu_id = 5;
			}else{
				printf("The BusID (%s) did not map correctly to a GPU. Exiting...\n", busid);
				exit(0);
			}

			if(strcmp(output_flag, "verbose") == 0){
				printf("MPI Rank %03d of %03d on HWThread %03d of Node %s - GPU %d of %d UUID: %s BusID: %s gpu_id: %d\n", rank, size, hwthread, name, i, num_devices, uuid, busid, gpu_id);
			}
			else{
				printf("MPI Rank %03d of %03d on HWThread %03d of Node %s - GPU %d of %d gpu_id: %d\n", rank, size, hwthread, name, i, num_devices, gpu_id);
			}

		}	

		nvmlShutdown();

	}

	MPI_Finalize();

	return 0;
}
